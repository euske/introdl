/* -*- mode: c -*- */

#include <hip/hip_runtime.h>
#include <stdio.h>

/* GPU で(並列に)実行される関数 */
__global__ void mult(float* out, float* a, float* b, int n)
{
    for (int i = 0; i < n; i++) {
        out[i] = a[i] * b[i];
    }
}

int main(int argc, char* argv[])
{
    int n = 1000000;

    /* データを用意する。 */
    float* a = (float*)malloc(sizeof(float) * n);
    float* b = (float*)malloc(sizeof(float) * n);
    float* out = (float*)malloc(sizeof(float) * n);
    for (int i = 0; i < n; i++) {
        a[i] = b[i] = i;
    }

    /* GPU上にデータ領域を割り当てる。 */
    float* c_a;
    float* c_b;
    float* c_out;
    hipMalloc(&c_a, sizeof(float) * n);
    hipMalloc(&c_b, sizeof(float) * n);
    hipMalloc(&c_out, sizeof(float) * n);

    /* CPU→GPU にデータを転送する。 */
    hipMemcpy(c_a, a, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(c_b, b, sizeof(float) * n, hipMemcpyHostToDevice);

    /* GPU 上で関数を実行する。 */
    mult<<<1, 1>>>(c_out, c_a, c_b, n);

    /* 計算結果を GPU→CPU に転送する。 */
    hipMemcpy(out, c_out, sizeof(float) * n, hipMemcpyDeviceToHost);

    /* 計算結果を表示する。 */
    printf("out[0]=%f\n", out[0]);
    printf("out[n-1]=%f\n", out[n-1]);

    /* 領域を開放。 */
    hipFree(c_a);
    hipFree(c_b);
    hipFree(c_out);
    free(a);
    free(b);
    free(out);
    return 0;
}
